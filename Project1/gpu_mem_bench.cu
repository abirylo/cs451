//gpu_bench.cu


#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
	    hipGetErrorString(err),__FILE__,__LINE__);	\
    exit(-1);						\
  }                                                     \


unsigned long MAX_OPS = 20000000;
const long MEGABYTE = 1048576;

__global__ void gpu_iops(unsigned long max_ops) {

//  int a = blockDim.x * blockIdx.x + threadIdx.x;
    
}


int main(int argc, char *argv[]) {
  
    char c;  
    char test = 'B';
    char rw = 'R';
    while ( (c = getopt(argc, argv, "r:t:") ) != -1)
    {
        switch (c) 
        {
            case 'r':
                rw = optarg[0];
                break;
            case 't':
                test = optarg[0];
                break;
            default:
                printf("Usage: ./benchCPU -n [number of threads]\n");
                return -1;
        }
    }
    struct timeval tv;
    long long start, stop;
    double secs;

    hipError_t err;
   
/*  
    err = cudaMalloc((void **) &d_string_array, sizeof(char**)*array_size); 
    CHECK_ERR(err); 

    err = cudaMalloc((void **) &d_answer_array, sizeof(bool)*array_size);
    CHECK_ERR(err);
      
    d_answer_array_copy = d_answer_array;

    err = cudaMalloc((void **) &d_command, line_size);
    CHECK_ERR(err);  

    err = cudaMalloc((void **) &d_contents, array_size*line_size);
    CHECK_ERR(err);
      
    err = cudaMemcpy(d_command, argv[1], strlen(argv[1])+1, cudaMemcpyHostToDevice);
    CHECK_ERR(err);      
*/
    unsigned char *d_mem_pointer;
    unsigned char *mem_pointer;
    if(test == 'B')
    {
      
      err = hipMalloc((void **) &d_mem_pointer, sizeof(unsigned char)*MEGABYTE);
      CHECK_ERR(err);
      mem_pointer = (unsigned char *)malloc(sizeof(unsigned char)*1);
      gettimeofday(&tv, NULL);
      start = tv.tv_sec*1000000LL + tv.tv_usec;
    
      for(unsigned long i = 0; i<MEGABYTE; i++)
      {
        err = hipMemcpy((void *)&d_mem_pointer[i], (void *)mem_pointer, 1, hipMemcpyHostToDevice);
        CHECK_ERR(err);
      }
      
      gettimeofday(&tv, NULL);
      stop = tv.tv_sec*1000000LL + tv.tv_usec;
      secs = (stop-start)/1000000.0;
      printf("Time taken: %lf\n", secs);
      printf("%lf MB/sec\n", 1.0/(secs)); 
    }
    else if(test == 'K')
    {
      err = hipMalloc((void **) &d_mem_pointer, sizeof(unsigned char)*256*MEGABYTE);
      CHECK_ERR(err);
      mem_pointer = (unsigned char *)malloc(sizeof(unsigned char)*1024);
      gettimeofday(&tv, NULL);
      start = tv.tv_sec*1000000LL + tv.tv_usec;
    
      for(unsigned long i = 0; i<256*MEGABYTE/1024; i++)
      {
        err = hipMemcpy((void *)&d_mem_pointer[i*1024], (void *)mem_pointer, 1024, hipMemcpyHostToDevice);
        CHECK_ERR(err);
      }
      
      gettimeofday(&tv, NULL);
      stop = tv.tv_sec*1000000LL + tv.tv_usec;
      secs = (stop-start)/1000000.0;
      printf("Time taken: %lf\n", secs);
      printf("%lf MB/sec\n", (256.0/1024.0)/(secs)); 
    }
    else if(test == 'M')
    {
      err = hipMalloc((void **) &d_mem_pointer, sizeof(unsigned char)*512*MEGABYTE);
      CHECK_ERR(err);
      mem_pointer = (unsigned char *)malloc(sizeof(unsigned char)*MEGABYTE);
      gettimeofday(&tv, NULL);
      start = tv.tv_sec*1000000LL + tv.tv_usec;
    
      for(unsigned long i = 0; i<512*10; i++)
      {
        err = hipMemcpy((void *)&d_mem_pointer[(i*MEGABYTE)%(512*MEGABYTE)], (void *)mem_pointer, MEGABYTE, hipMemcpyHostToDevice);
        CHECK_ERR(err);
      }
      
      gettimeofday(&tv, NULL);
      stop = tv.tv_sec*1000000LL + tv.tv_usec;
      secs = (stop-start)/1000000.0;
      printf("Time taken: %lf\n", secs);
      printf("%lf MB/sec\n", (512*10)/(secs)); 
    }
    /*err = cudaFree(d_contents);
    CHECK_ERR(err);
    err = cudaFree(d_string_array);
    CHECK_ERR(err);
    err = cudaFree(d_answer_array);
    CHECK_ERR(err);
 */
}
