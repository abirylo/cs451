#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <string.h>
#include "hashtable.cuh"
#include "hashtable.h"

#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
      hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);           \
  }                                                     \

#define MEMORY_SIZE 268435456 //256Megabytes  
#define BLOCK_SIZE 128
  
__global__ void wordcount(char* words, char* result, long* size_result, long size, int n) {

  int a = blockDim.x * blockIdx.x + threadIdx.x;
  
  if(a < n){
    //find start pointer offset
    char* start = (MEMORY_SIZE/BLOCK_SIZE) * (a) + words; 
    char* end = (MEMORY_SIZE/BLOCK_SIZE) * (a+1) + words;
    if(end > words+size)
      end = words+size;
    
    hashtable_t table;
    ht_init_s(&table, 1024);
    
// "?\";<>,~`!@#^&*()_+-=/\\:;{}[]|. "
    char *word_start = start;
    char *current_pos = start;
    while(current_pos <= end){
      if(*current_pos == '?' || *current_pos == '\"' || *current_pos == ';' || *current_pos == '<' || *current_pos == '>' || *current_pos == ',' || *current_pos == '~' || *current_pos == '`' || *current_pos == '!' || *current_pos == '@' || *current_pos == '#' || *current_pos == '^' || *current_pos == '&' || *current_pos == '*' || *current_pos == '(' || *current_pos == ')' || *current_pos == '_' || *current_pos == '+' || *current_pos == '-' || *current_pos == '=' || *current_pos == '/' || *current_pos == '\\' || *current_pos == ':' || *current_pos == '{' || *current_pos == '}' || *current_pos == '[' || *current_pos == ']' || *current_pos == '|' || *current_pos == '.' || *current_pos == ' ' || *current_pos == '\n' || *current_pos == '\0'){
        *current_pos = '\0';
        int val; 
        if((val = ht_get(&table, word_start)) != -1){
          ht_delete(&table, word_start);
          ht_add(&table, word_start, val+1);
        }
        else{
          ht_add(&table, word_start, 1);
        }
        word_start = current_pos+1;
      }
      current_pos++;
    }
       
    char *r = (MEMORY_SIZE/BLOCK_SIZE) * a + result;
    get_all_kvps(&table, (kvp_t*)r);
    size_result[a] = table.size;
    
    ht_dispose(&table);
    //__syncthreads();
  
  }
}


int main(int argc, char *argv[]) {

  hipError_t err;
  
  char* h_words;
  char* d_words;
  char* h_result;
  char* d_result;
  long* h_size_result;
  long* d_size_result;
  //char** h_dout;
  //char** d_dout;
  char* file_name = argv[1];
  FILE* file;
  long file_size = 0;
  long pos = 0;
  struct stat64 st;
  //char* contents;

  stat64(file_name, &st);
  file_size = st.st_size;

  file = fopen(file_name, "r");
  if(file == NULL)
    return -1;
  
  hashtable_t table;
  ht_init_s(&table, 1024*1024);
  
  h_words = (char*)malloc(sizeof(char)*(MEMORY_SIZE>file_size ? file_size : MEMORY_SIZE)); 
  err = hipMalloc((void **) &d_words, sizeof(char)*MEMORY_SIZE);
  CHECK_ERR(err);

  h_result = (char*)malloc(sizeof(char)*MEMORY_SIZE);
  err = hipMalloc((void **) &d_result, sizeof(char)*MEMORY_SIZE);
  CHECK_ERR(err);

  h_size_result = (long *)malloc(sizeof(long)*BLOCK_SIZE);
  err = hipMalloc((void **) &d_size_result, sizeof(long)*BLOCK_SIZE);
  CHECK_ERR(err);

  while(pos < file_size){
    long read_size = read(file->_fileno, h_words, MEMORY_SIZE);
    pos += read_size;
    
    err = hipMemcpy(d_words, h_words, read_size, hipMemcpyHostToDevice);
    CHECK_ERR(err);
   
    int threads = (int)ceil((double)(read_size*BLOCK_SIZE)/(double)MEMORY_SIZE); 
    wordcount<<< 1, BLOCK_SIZE >>>(d_words,d_result,d_size_result,read_size,threads); 
    
    err = hipMemcpy(h_words, d_words, read_size, hipMemcpyDeviceToHost);
    CHECK_ERR(err);
    err = hipMemcpy(h_result, d_result, threads*MEMORY_SIZE/BLOCK_SIZE, hipMemcpyDeviceToHost);
    CHECK_ERR(err);
    err = hipMemcpy(h_size_result, d_size_result, BLOCK_SIZE, hipMemcpyDeviceToHost);
    CHECK_ERR(err);
    
    //need to merge
    for(int i=0; i<threads, i++){
      for(int j=0; i<h_size_result[i]; j++)
        kvp_t *kvps = (kvp_t*)h_result;
        char * c = (char *)(kvps[i].key-(long)d_words)+(long)h_words;
        if((val = ht_get(&table, c)) != -1){
          ht_add(&table, word_start, val+1);
        }
        else{
          char* s = malloc(strlen(word_start);
          strcpy(s, word_start);
          ht_add(&table, s, 1);
        }
      }
    }
  }

  err = hipFree(d_words);
  CHECK_ERR(err);
  err = hipFree(d_result);
  CHECK_ERR(err);
  err = hipFree(d_size_result);
  CHECK_ERR(err);
  
  free(h_words);
  free(h_result);
  free(h_size_result);
  //print out the final hash table
  kvp_t* results = malloc(sizeof(kvp_t)*table.size);
  get_all_kvps(&table, (kvp_t*)r);
  
  for(int i=0; i<table.size; i++){
    print("%s %i\n",r[i].key, r[i].val);
    free(r[i].key);
  }
  
  ht_dispose(&table);
  free(r);

}
