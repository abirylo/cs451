#include "hip/hip_runtime.h"
/*
 * Adrian Birylo
 * abirylo@iit.edu
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <string.h>


#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
	    hipGetErrorString(err),__FILE__,__LINE__);	\
    exit(-1);						\
  }                                                     \

__global__ void gpu() {

  int a = blockDim.x * blockIdx.x + threadIdx.x;

  if(a <= n)
  {
    d_out[a] = false;
    bool match = false;
    int i=0;
    int j=0;
  }
}

int main(int argc, char *argv[]) {
  
    hipError_t err;
   
/*  
    err = hipMalloc((void **) &d_string_array, sizeof(char**)*array_size); 
    CHECK_ERR(err); 

    err = hipMalloc((void **) &d_answer_array, sizeof(bool)*array_size);
    CHECK_ERR(err);
      
    d_answer_array_copy = d_answer_array;

    err = hipMalloc((void **) &d_command, line_size);
    CHECK_ERR(err);  

    err = hipMalloc((void **) &d_contents, array_size*line_size);
    CHECK_ERR(err);
      
    err = hipMemcpy(d_command, argv[1], strlen(argv[1])+1, hipMemcpyHostToDevice);
    CHECK_ERR(err);      
*/
    CUdevprop prop = malloc(sizeof(CUdevprop));
             //run grep    
    gpu<<< ceil(array_size/1024), 1024 >>>();        
 

 /*
    err = hipFree(d_contents);
    CHECK_ERR(err);
    err = hipFree(d_string_array);
    CHECK_ERR(err);
    err = hipFree(d_answer_array);
    CHECK_ERR(err);
 */
  }
}
